#include "hip/hip_runtime.h"
#include "globalDefs.h"
#include "hook_kernels.h"
#define NTYPES 3
__global__ void copyToBuffer_cu(float4 *xs, float4 *vs, uint *ids, int *idToIdxs, char *buffer, uint *idsToCopy, int n) {
    int tid = GETIDX();
    if (tid < n * NTYPES) {
        int dataType = tid / n;
        int baseIdx = dataType * n;
        int nAtom = tid - baseIdx;
        int id = idsToCopy[nAtom];
        int idx = idToIdxs[id];
       // printf("tid %d dtype %d nAtom %d id %d idx %d\n", tid, dataType, nAtom, id, idx);
        if (dataType == 0) {
            float4 *bFloat = (float4 *) buffer;
            bFloat[nAtom] = xs[idx];
        } else if (dataType == 1) {
            float4 *bFloat = ((float4 *) buffer) + n;
            bFloat[nAtom] = vs[idx];
        } else if (dataType == 2) {
            uint *bInt = (uint *) (((float4 *) buffer) + 2*n);
            bInt[nAtom] = id; 
        }

//        int idx = 
    }
}

__global__ void unpackBuffer_cu(float4 *fs, int *idToIdxs, float4 *biasForces, uint *idsToCopy, int n) {
    int tid = GETIDX();
    if (tid < n) {
        int idx = idToIdxs[idsToCopy[tid]];
        float4 biasForce = biasForces[tid];
        float4 f = fs[idx];
        f.x += biasForce.x;
        f.y += biasForce.y;
        f.z += biasForce.z;
        fs[idx] = f;
    }
}


void copyToBuffer(float4 *xs, float4 *vs, uint *ids, int *idToIdxs, char *buffer, uint *idsToCopy, int n) {
    copyToBuffer_cu<<<NBLOCK(NTYPES * n), PERBLOCK>>>(xs, vs, ids, idToIdxs, buffer, idsToCopy, n);

}



void unpackBuffer(float4 *fs, int *idToIdxs, float4 *biasForces, uint *idsToCopy, int n) {
    unpackBuffer_cu<<<NBLOCK(n), PERBLOCK>>>(fs, idToIdxs, biasForces, idsToCopy, n);
}
