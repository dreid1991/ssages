#include "hip/hip_runtime.h"
#include "globalDefs.h"
#include "cutils_math.h"

__global__ void umbrella_eval(float4 *fs, float *val, float4 *grad, float center, float k, int nAtoms) {
    int idx = GETIDX();
    if (idx < nAtoms) {
        float v = val[0];
        float4 f = fs[idx];
        float wOrig = f.w;
        float4 g = grad[idx];
        float mag = k * (v - center);
        g = g * mag;
        f -= g;
        //printf("g is %f %f %f %f\n", g.x, g.y, g.z, g.w);
        f.w = wOrig;
        fs[idx] = f;
    }
}

void call_umbrella_eval(float4 *fs, float *val, float4 *grad, float center, float k, int nAtoms) {
    umbrella_eval<<<NBLOCK(nAtoms), PERBLOCK>>>(fs, val, grad, center, k, nAtoms);

}
