#include "hip/hip_runtime.h"
#include "globalDefs.h"
#include "cutils_math.h"

__global__ void umbrella_eval(float4 *fs, float *val, float4 *grad, float center, float k, int nAtoms) {
    int idx = GETIDX();
    if (idx < nAtoms) {
        float v = val[0];
        printf("from kernel, v is %f\n", v);
        float4 f = fs[idx];
        printf("idx is %d\n", idx);
        printf("fw is %f\n", f.w);
        float wOrig = f.w;
        float4 g = grad[idx];
        printf("g is %f\n", g.x);
        float mag = k * (v - center);
        printf("v is %f, center is %f, mag is %f\n", v, center, mag);
        g = g * mag;
        f -= g;
        f.w = wOrig;
        fs[idx] = f;
    }
}

void call_umbrella_eval(float4 *fs, float *val, float4 *grad, float center, float k, int nAtoms) {
    SAFECALL((umbrella_eval<<<NBLOCK(nAtoms), PERBLOCK>>>(fs, val, grad, center, k, nAtoms)));

}
