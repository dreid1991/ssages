#include "hip/hip_runtime.h"
#include "AtomCoordinateCV_gpu_kernels.h"
#include "globalDefs.h"//from DANMD


__global__ void call_atom_coordinate_eval_cu(float4 *xs, int *idToIdxs, float *val, float4 *grad, int atomId, int index, int nAtoms) {
    int idx = GETIDX();
    if (idx < nAtoms) {
        int id = idToIdxs[idx];
        float4 atomGrad = make_float4(0, 0, 0, 0);
        if (id == atomId) {
            if (index == 0) {
                atomGrad.x = 1;
                val[0] = xs[idx].x;
            } else if (index == 1) {
                atomGrad.y = 1;
                val[0] = xs[idx].y;
            } else if (index == 2) {
                atomGrad.z = 1;
                val[0] = xs[idx].z;
            }
        }
        grad[idx] = atomGrad;

    }
}
void call_atom_coordinate_eval(float4 *xs, int *idToIdxs, float *val, float4 *grad, int atomId, int index, int nAtoms) {
    SAFECALL((call_atom_coordinate_eval_cu<<<NBLOCK(nAtoms), PERBLOCK>>>(xs, idToIdxs, val, grad, atomId, index, nAtoms)));
}
