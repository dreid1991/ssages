#include "hip/hip_runtime.h"
#include "AtomCoordinateCV_gpu_kernels.h"
#include "globalDefs.h"//from DANMD


__global__ void call_atom_coordinate_eval_cu(float4 *xs, uint *ids, float *val, float4 *grad, int atomId, int index, int nAtoms) {
    int idx = GETIDX();
    if (idx < nAtoms) {
        int id = ids[idx];
        float4 atomGrad = make_float4(0, 0, 0, 0);
        if (id == atomId) {
            if (index == 0) {
                atomGrad.x = 1;
                val[0] = xs[idx].x;
            } else if (index == 1) {
                atomGrad.y = 1;
                val[0] = xs[idx].y;
            } else if (index == 2) {
                atomGrad.z = 1;
                val[0] = xs[idx].z;
            }
        }
        grad[idx] = atomGrad;

    }
}
void call_atom_coordinate_eval(float4 *xs, uint *ids, float *val, float4 *grad, int atomId, int index, int nAtoms) {
    call_atom_coordinate_eval_cu<<<NBLOCK(nAtoms), PERBLOCK>>>(xs, ids, val, grad, atomId, index, nAtoms);
}
