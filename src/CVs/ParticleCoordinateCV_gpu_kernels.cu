#include "hip/hip_runtime.h"
#include "ParticleCoordinateCV_gpu_kernels.h"
#include "globalDefs.h"//from DANMD
#include "cutils_func.h"


__global__ void call_particle_position_mass_eval_cu(float4 *xs,float4 *vs, BoundsGPU bounds, int *idToIdxs, float4 *buf, int* atomIds,int atomIdsize, int nAtoms) {
    int idx = GETIDX();
    if (idx < atomIdsize) {
        int id = atomIds[idx];
        int Idx= idToIdxs[id];
        float4 x=xs[Idx];
        float3 pos = make_float3(x);
        //get first atom postion
        float3 pos0 = make_float3(xs[idToIdxs[atomIds[0]]]);
        float3 dr = bounds.minImage(pos - pos0)+pos0;
        float massinv=vs[Idx].w;
        x=make_float4(dr);
        x.w=massinv;
        buf[idx] = x;
        
    }
}

__global__ void call_grad_eval_cu(float4 *vs, int *idToIdxs, float4 *grad, int* atomIds,int atomIdsize,float4 *totalmass, float3 index) {
    int idx = GETIDX();
    if (idx < atomIdsize) {
        int id = atomIds[idx];
        int Idx= idToIdxs[id];
        float massinv=vs[Idx].w;
        float4 atomGrad = make_float4(index)/(massinv*totalmass[0].w);
        grad[Idx] = atomGrad;
//          printf("atomGrad  %f %f %f  %f %d\n", atomGrad.x,atomGrad.y,atomGrad.z,atomGrad.w,id);

    }
}

__global__ void call_wrap_particle_coordinate_eval_cu(BoundsGPU bounds, float *val,float4 *sum_buf, float3 index) {
    int idx = GETIDX();
    if (idx < 1) {
        float4 posmass=sum_buf[0];
        float3 pos=make_float3(posmass)/posmass.w;

//         pos = bounds.wrap(pos);
        float3 trace = bounds.trace();
        float3 diffFromLo = pos - bounds.lo;
        float3 imgs = floorf(diffFromLo / trace); //are unskewed at this point
        pos -= trace * imgs * bounds.periodic;
        float a=dot(pos,index);

        val[0]=a;

    }
}



void call_particle_position_mass_eval(float4 *xs,float4 *vs,BoundsGPU boundsGPU, int *idToIdxs, float4 *buf,float4 *sum_buf,float* val, int* atomIds,int atomIdsize, float3 index, int nAtoms, int warpize)
{
  

    SAFECALL((call_particle_position_mass_eval_cu<<<NBLOCK(atomIdsize), PERBLOCK>>>(xs, vs,boundsGPU,idToIdxs, buf, atomIds, atomIdsize, nAtoms)));

    SAFECALL((accumulate_gpu<float4,float4, SumVectorXYZOverW, N_DATA_PER_THREAD> <<<NBLOCK(atomIdsize/(double)N_DATA_PER_THREAD),PERBLOCK,N_DATA_PER_THREAD*sizeof(float4)*PERBLOCK>>>
            (
             sum_buf,
             buf,
             atomIdsize,
             warpize,
             SumVectorXYZOverW()
             )));    
    
     SAFECALL((call_wrap_particle_coordinate_eval_cu<<<NBLOCK(1), PERBLOCK>>>(boundsGPU,val, sum_buf,index)));
}


 void call_grad_eval(float4 *vs, int *idToIdxs, float4 *grad, int* atomIds,int atomIdsize,float4 *totalmass, float3 index) {
      SAFECALL((call_grad_eval_cu<<<NBLOCK(atomIdsize), PERBLOCK>>>( vs,idToIdxs, grad, atomIds, atomIdsize, totalmass,index)));
 }


__global__ void call_particle_coordinate_eval_cu(float4 *xs,float4 *vs, BoundsGPU bounds, int *idToIdxs, float *buf, float4 *grad, int* atomIds,int atomIdsize,float massinv, int index, int nAtoms) {
    int idx = GETIDX();
    if (idx < atomIdsize) {
        int id = atomIds[idx];
        int Idx= idToIdxs[id];
        float4 atomGrad = make_float4(0, 0, 0, 0);
        float tval=0;
        float4 x=xs[Idx];
        float3 pos = make_float3(x);
        //get first atom postion
        float3 pos0 = make_float3(xs[idToIdxs[atomIds[0]]]);
        float3 dr = bounds.minImage(pos - pos0)+pos0;
        float4 v=vs[Idx];
        if (index == 0) {
            atomGrad.x = 1;
            tval = dr.x;
        } else if (index == 1) {
            atomGrad.y = 1;
            tval = dr.y;
        } else if (index == 2) {
            atomGrad.z = 1;
            tval = dr.z;
        }
        grad[Idx] = atomGrad/v.w*massinv;
        buf[idx] = tval/v.w*massinv;//TODO use buf value or do not calc mass every timestep
//  printf("buf   %f %d\n", tval,id);
    }
}


void call_particle_coordinate_eval(float4 *xs,float4 *vs,BoundsGPU boundsGPU, int *idToIdxs, float *buf, float4 *grad, int* atomIds,int atomIdsize,float *val,float massinv, int index, int nAtoms, int warpize) {
  

    SAFECALL((call_particle_coordinate_eval_cu<<<NBLOCK(atomIdsize), PERBLOCK>>>(xs, vs,boundsGPU,idToIdxs, buf, grad, atomIds, atomIdsize,massinv, index, nAtoms)));

    SAFECALL((accumulate_gpu<float,float, SumSingle, N_DATA_PER_THREAD> <<<NBLOCK(atomIdsize/(double)N_DATA_PER_THREAD),PERBLOCK,N_DATA_PER_THREAD*sizeof(float)*PERBLOCK>>>
            (
             val,
             buf,
             atomIdsize,
             warpize,
             SumSingle()
             )));     
}


__global__ void call_particle_mass_eval_cu(float4 *vs, int *idToIdxs, float *buf,  int* atomIds,int atomIdsize, int nAtoms) {
    int idx = GETIDX();
    if (idx < atomIdsize) {
        int id = atomIds[idx];
        int Idx= idToIdxs[id];
        buf[idx] = 1.0/vs[Idx].w;
    }
}


void call_particle_mass_eval(float4 *vs, int *idToIdxs, float *buf,int* atomIds, int atomIdsize, float *mass, int nAtoms, int warpSize) {

    SAFECALL((call_particle_mass_eval_cu<<<NBLOCK(atomIdsize), PERBLOCK>>>(vs, idToIdxs, buf,  atomIds, atomIdsize, nAtoms)));
    
    SAFECALL((accumulate_gpu<float,float, SumSingle, N_DATA_PER_THREAD> <<<NBLOCK(atomIdsize/(double)N_DATA_PER_THREAD),PERBLOCK,N_DATA_PER_THREAD*sizeof(float)*PERBLOCK>>>
            (
             mass,
             buf,
             atomIdsize,
             warpSize,
             SumSingle()
             )));     

}


__global__ void call_wrap_particle_coordinate_eval_cu(BoundsGPU bounds, float *val, float3 index) {
    int idx = GETIDX();
    if (idx < 1) {
        float a=val[0];
        float3 pos = index*a;
//         pos = bounds.wrap(pos);
        float3 trace = bounds.trace();
        float3 diffFromLo = pos - bounds.lo;
        float3 imgs = floorf(diffFromLo / trace); //are unskewed at this point
        pos -= trace * imgs * bounds.periodic;
        if (imgs.x != 0 or imgs.y != 0 or imgs.z != 0) {
            a=dot(pos,index);
            val[0]=a;
        }
    }
}


void call_wrap_particle_coordinate_eval( BoundsGPU boundsGPU, float *val, float3 index){

    
      SAFECALL((call_wrap_particle_coordinate_eval_cu<<<NBLOCK(1), PERBLOCK>>>(boundsGPU,val, index)));
  
}
