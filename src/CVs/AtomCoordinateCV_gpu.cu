#include "hip/hip_runtime.h"
#include "AtomCoordinateCV_gpu.h"
#include "globalDefs.h"//from DANMD

__global__ void eval_cu(float4 *xs, int *idToIdxs, float *val, float4 *grad, int atomId, int index, int nAtoms) {
    int idx = GETIDX();
    if (idx < nAtoms) {
        int id = idToIdxs[idx];
        float3 atomGrad = make_float4(0, 0, 0, 0);
        if (id == atomId) {
            if (index == 0) {
                atomGrad.x = 1;
                val[0] = xs[idx].x;
            } else if (index == 1) {
                atomGrad.y = 1;
                val[0] = xs[idx].y;
            } else if (index == 2) {
                atomGrad.z = 1;
                val[0] = xs[idx].z;
            }
        }
        grad[idx] = atomGrad;

    }
}

void AtomCoordinateCV_gpu::Evaluate(const Snapshot& snapshot) 
{
    // Gradient and value. 
    float4 *xs = snapshot._gpd.xs;
    int *idToIdxs = snapshot._gpd.idToIdxs;
    const auto& pos = snapshot.GetPositions(); 
    const auto& ids = snapshot.GetAtomIDs();
    int nAtoms = snapshot._gpd.nAtoms;
    // Loop through atom positions.
    eval_cu<<<NBLOCK(nAtoms), PERBLOCK>>>(xs, idToIdxs, _val, _grad, _atomId, _index, nAtoms)

}


